#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"

struct MinimumKernelOp {};

LONG_BINARY_OP(float, minimum_forward_f32, minimum_backward_f32, MinimumKernelOp,
    {
        fx = fminf(x, y);
    },
    {
        if (x < y) {
            dfdx = 1.0;
            dfdy = 0.0;
        } else if (x > y) {
            dfdx = 0.0;
            dfdy = 1.0;
        } else {
            dfdx = 0.5;
            dfdy = 0.5;
        }
    }
)
