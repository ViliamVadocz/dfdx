#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"

template<typename F>
struct HuberErrorOp {
    F delta;
};

LONG_BINARY_OP(float, huber_error_forward_f32, huber_error_backward_f32, HuberErrorOp<float>,
    {
        float a = x - y;

        if (fabsf(a) < op.delta) {
            fx = a * a * 0.5;
        } else {
            fx = op.delta * (fabsf(a) - 0.5 * op.delta);
        }
    },
    {
        auto a = x - y;

        if (a == 0.0) {
            dfdx = 0.0;
        } else if (fabsf(a) < op.delta) {
            dfdx = a;
        } else {
            dfdx = copysignf(op.delta, a);
        }

        dfdy = -dfdx;
    }
)

LONG_BINARY_OP(double, huber_error_forward_f64, huber_error_backward_f64, HuberErrorOp<double>,
    {
        double a = x - y;

        if (fabs(a) < op.delta) {
            fx = a * a * 0.5;
        } else {
            fx = op.delta * (fabs(a) - 0.5 * op.delta);
        }
    },
    {
        auto a = x - y;

        if (a == 0.0) {
            dfdx = 0.0;
        } else if (fabs(a) < op.delta) {
            dfdx = a;
        } else {
            dfdx = copysign(op.delta, a);
        }

        dfdy = -dfdx;
    }
)
