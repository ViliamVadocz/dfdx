#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

template<typename F>
struct PowFKernelOp {
    F rhs;
};

UNARY_OP(__half, pow_fwd_f16, pow_bwd_f16, PowFKernelOp<__half>,
    powf(x, op.rhs),
    op.rhs * powf(x, op.rhs - 1.0))

UNARY_OP(float, pow_fwd_f32, pow_bwd_f32, PowFKernelOp<float>,
        powf(x, op.rhs),
        op.rhs * powf(x, op.rhs - 1.0))

UNARY_OP(double, pow_fwd_f64, pow_bwd_f64, PowFKernelOp<double>,
    pow(x, op.rhs),
    op.rhs * pow(x, op.rhs - 1.0))
    