#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

template<typename F>
struct NansToKernelOp {
    F x;
};

UNARY_OP(float, nans_to_forward_f32, nans_to_backward_f32, NansToKernelOp<float>,
        isnan(x) ? op.x : x,
        isnan(x) ? 0.0 : 1.0)
