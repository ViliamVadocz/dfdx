#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct CosKernelOp {};

UNARY_OP(__half, cos_fwd_f16, cos_bwd_f16, CosKernelOp,
        cosf(x),
        -sinf(x))

UNARY_OP(float, cos_fwd_f32, cos_bwd_f32, CosKernelOp,
        cosf(x),
        -sinf(x))

UNARY_OP(double, cos_fwd_f64, cos_bwd_f64, CosKernelOp,
        cos(x),
        -sin(x))
