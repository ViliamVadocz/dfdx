#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

#define SIGMOID_f32(X) (1.0 / (1.0 + expf(-X))) 
#define SIGMOID_f64(X) (1.0 / (1.0 + exp(-X))) 

struct SigmoidKernelOp {};

UNARY_OP(float, sigmoid_forward_f32, sigmoid_backward_f32, SigmoidKernelOp,
        SIGMOID_f32(x),
        SIGMOID_f32(x) * (1.0 - SIGMOID_f32(x)))

UNARY_OP(double, sigmoid_forward_f64, sigmoid_backward_f64, SigmoidKernelOp,
        SIGMOID_f64(x),
        SIGMOID_f64(x) * (1.0 - SIGMOID_f64(x)))
        