#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct SqrtKernelOp {};

UNARY_OP(__half, sqrt_fwd_f16, sqrt_bwd_f16, SqrtKernelOp,
        sqrtf(x),
        1 / (y + y))

UNARY_OP(float, sqrt_fwd_f32, sqrt_bwd_f32, SqrtKernelOp,
        sqrtf(x),
        1 / (y + y))

UNARY_OP(double, sqrt_fwd_f64, sqrt_bwd_f64, SqrtKernelOp,
        sqrt(x),
        1 / (y + y))
        