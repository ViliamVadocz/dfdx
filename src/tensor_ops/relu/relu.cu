#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct ReLUKernelOp {};

UNARY_OP(float, relu_forward_f32, relu_backward_f32, ReLUKernelOp,
        fmaxf(x, 0.0),
        x > 0.0 ? 1.0 : 0.0)
