#include "hip/hip_runtime.h"
#include "unary_op_macros.cuh"

struct ExpKernelOp {};

UNARY_OP(float, exp_forward_f32, exp_backward_f32, ExpKernelOp,
        expf(x),
        expf(x))
